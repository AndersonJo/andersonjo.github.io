
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (1024*33)

__global__ void add(int *a, int *b, int *result){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while(tid<N){
		result[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
	printf("gridDim.x: %d, blockIdx.x: %d, threadIdx.x: %d, blockDim.x: %d\n",gridDim.x, blockIdx.x, threadIdx.x, blockDim.x);
}

void deviceInfo(){
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
	printf("maxGridSize: %p\n", prop.maxGridSize[0]);
	printf("maxGridSize: %d\n", prop.maxGridSize[1]);
	printf("maxGridSize: %d\n", prop.maxGridSize[2]);
}

int main(int argc, char **argv) {
	int a[N];
	int b[N];
	int c[N];
	int *dev_a;
	int *dev_b;
	int *dev_c;

	deviceInfo();



	for(int i=0; i<N; i++){
		a[i] = i;
		b[i] = i*2;
	}

	hipMalloc((void**)&dev_a, sizeof(int)*N);
	hipMalloc((void**)&dev_b, sizeof(int)*N);
	hipMalloc((void**)&dev_c, sizeof(int)*N);

	hipMemcpy(dev_a, a, sizeof(int)*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(int)*N, hipMemcpyHostToDevice);

	add<<<1, 10>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, sizeof(int)*N, hipMemcpyDeviceToHost);

	double result =0;
	for(int i=0; i<N; i++){
//		printf("%d + %d = %d\n", a[i], b[i], c[i]);
		result += c[i];
	}
	printf("result: %f", result);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
